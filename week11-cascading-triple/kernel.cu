﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



void print_array(float* A, int N)
{
    for (int i = 0; i < N; i++)
        printf("%.2f ", A[i]);
    printf("\n");
}


__global__ void
compute_kernel1(float* input1, float* input2, float* output, int datasize)
{
    int numElements = datasize / sizeof(float);

    // Write code for i
    int innerThreadId = threadIdx.x + (threadIdx.y * blockDim.x) + (threadIdx.z * (blockDim.x * blockDim.y));
    int globalBlockId = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * (gridDim.x * gridDim.y));
    int i = (globalBlockId * (blockDim.x * blockDim.y * blockDim.z)) + innerThreadId;

    if (i < numElements)
    {
        //Write code for compute
        output[i] = (input1[i] * input2[i]) * 4;
    }
}


__global__ void
compute_kernel2(float* input, float* output, int datasize)
{
    int numElements = datasize / sizeof(float);

    // Write code for i
    int innerThreadId = threadIdx.x + (threadIdx.y * blockDim.x) + (threadIdx.z * (blockDim.x * blockDim.y));
    int globalBlockId = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * (gridDim.x * gridDim.y));
    int i = (globalBlockId * (blockDim.x * blockDim.y * blockDim.z)) + innerThreadId;

    if (i < numElements)
    {
        //Write code for compute
        output[i] = input[i] * input[i];
    }
}


__global__ void
compute_kernel3(float* input, float* output, int datasize)
{
    int numElements = datasize / sizeof(float);

    // Write code for i
    int innerThreadId = threadIdx.x + (threadIdx.y * blockDim.x) + (threadIdx.z * (blockDim.x * blockDim.y));
    int globalBlockId = blockIdx.x + (blockIdx.y * gridDim.x) + (blockIdx.z * (gridDim.x * gridDim.y));
    int i = (globalBlockId * (blockDim.x * blockDim.y * blockDim.z)) + innerThreadId;

    if (i < numElements)
    {
        output[i] = input[i] * i;
        //Write code for compute
    }
}



int main(void)
{
    hipError_t err = hipSuccess;

    int numElements = 16384;
    size_t size = numElements * sizeof(float);

    float* h_input1 = (float*)malloc(size);

    float* h_input2 = (float*)malloc(size);

    float* h_output1 = (float*)malloc(size);

    float* h_output2 = (float*)malloc(size);

    float* h_output3 = (float*)malloc(size);

    if (h_input1 == NULL || h_input2 == NULL || h_output1 == NULL || h_output2 == NULL || h_output3 == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }



    for (int i = 0; i < numElements; ++i)
    {
        scanf("%f", &h_input1[i]);

    }
    for (int i = 0; i < numElements; ++i)
    {
        scanf("%f", &h_input2[i]);

    }



    float* d_input1 = NULL;
    err = hipMalloc((void**)&d_input1, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_input1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float* d_input2 = NULL;
    err = hipMalloc((void**)&d_input2, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_input2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float* d_output1 = NULL;
    err = hipMalloc((void**)&d_output1, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector h_output1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float* d_output2 = NULL;
    err = hipMalloc((void**)&d_output2, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector h_output2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float* d_output3 = NULL;
    err = hipMalloc((void**)&d_output3, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector h_output3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_input1, h_input1, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector h_input1 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_input2, h_input2, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector h_input2 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Complete Code for launching compute_kernel1
    dim3 k1_dimGrid(4, 2, 2);
    dim3 k1_dimBlock(32, 32, 1);
    compute_kernel1<<<k1_dimGrid, k1_dimBlock>>>(d_input1, d_input2, d_output1, size);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel1 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Complete Code for launching compute_kernel2
    dim3 k2_dimGrid(2, 8, 1);
    dim3 k2_dimBlock(8, 8, 16);
    compute_kernel2<<<k2_dimGrid, k2_dimBlock>>>(d_output1, d_output2, size);
    
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel2 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Complete Code for launching compute_kernel3 
    dim3 k3_dimGrid(16, 1, 1);
    dim3 k3_dimBlock(128, 8, 1);
    compute_kernel3<<<k3_dimGrid, k3_dimBlock>>>(d_output2, d_output3, size);


    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel3 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    

    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_output1, d_output1, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output1 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_output2, d_output2, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output2 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_output3, d_output3, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output3 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }




    print_array(h_output3, numElements);


    err = hipFree(d_input1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_input2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(h_input1);
    free(h_input2);
    free(h_output1);
    free(h_output2);
    free(h_output3);

    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    return 0;
}

